
#include <hip/hip_runtime.h>
#include <math.h>

__global__
void add(int n, float *x, float *y)
{
    for(int i = 0; i < n; i++)
    {
        y[i] = x[i] + y[i];
    }
}

int main(void)
{
    int N = 1 << 20;
    float *x, *y;
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));
    for(int i = 0; i < N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    add<<<1, 1>>>(N, x, y);

    hipDeviceSynchronize();
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
    {
        maxError = fmax(maxError, fabs(3.0f - y[i]));
    }
    printf("Max Error: ");
    printf("%f", maxError);

    hipFree(x);
    hipFree(y);
    
    return 0;   
}